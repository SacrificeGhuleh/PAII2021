#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <timer.h>
#include <type_traits>
#include <benchmark.h>

#define COMP_TYPE int

constexpr size_t width = 10U; //cols
constexpr size_t height = 5U; // rows

constexpr size_t widthInBytes = width * sizeof(COMP_TYPE); //cols
constexpr size_t heightInBytes = height * sizeof(COMP_TYPE); // rows

constexpr size_t length = width * height;
constexpr size_t TPB = 8;

hipError_t err = hipError_t::hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();


template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void fill(T *__restrict__ arr, const size_t pitchInElements) {
  const size_t col = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t row = blockDim.x * blockIdx.x + threadIdx.x;
  
  const size_t idx = row * pitchInElements + col;

//  printf("--------------------------\n");
//  printf("col: %d, row: %d idx: %d\n", col, row, idx);
//  printf("bd: %d, %d %d\n", blockDim.x, blockDim.y, blockDim.z);
//  printf("bi: %d, %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
//  printf("ti: %d, %d %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
//  printf("--------------------------\n\n");

//  printf("--------------------------\ncol: %d, row: %d idx: %d\nbd: %d, %d %d\nbi: %d, %d %d\nti: %d, %d %d\n--------------------------\n\n",
//         col, row, idx,
//         blockDim.x, blockDim.y, blockDim.z,
//         blockIdx.x, blockIdx.y, blockIdx.z,
//         threadIdx.x, threadIdx.y, threadIdx.z);
//
  
  if ((row < height) && (col < width)) {
    arr[idx] = col * height + row;
  }
}


template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void increment(T *__restrict__ arr, const size_t pitchInElements) {
  const size_t col = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t row = blockDim.x * blockIdx.x + threadIdx.x;
  
  const size_t idx = row * pitchInElements + col;
  
  if ((row < height) && (col < width)) {
    arr[idx] = arr[idx] << 1;
//    arr[idx] += 1;
  }
}

int main() {
  initializeCUDA(deviceProp);
  
  COMP_TYPE *deviceArray = nullptr;
  size_t pitchInBytes = 0;
  checkCudaErrors(hipMallocPitch((void **) &deviceArray, &pitchInBytes, widthInBytes, height));
  const size_t pitchInElements = pitchInBytes / sizeof(COMP_TYPE);
  
  printf("Pitch: %zu B (%zu items)\n", pitchInBytes, pitchInElements);
  
  dim3 dimBlock(TPB, TPB, 1);
  dim3 dimGrid(getNumberOfParts(height, TPB), getNumberOfParts(width, TPB), 1); // Great number of blocks
  
  printf("Dim block: %d, %d, %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
  printf("Dim  grid: %d, %d, %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
  
  
  fill<<<dimGrid, dimBlock>>>(deviceArray, pitchInElements);
  
  checkDeviceMatrix(deviceArray, pitchInBytes, height, width, "\t%d", "Device filled data");
  
  increment<<<dimGrid, dimBlock>>>(deviceArray, pitchInElements);
  
  
  COMP_TYPE *hostArray = static_cast<COMP_TYPE *>(::operator new(pitchInBytes * height));
  
  hipMemcpy2D(hostArray, pitchInBytes, deviceArray, pitchInBytes, widthInBytes, height, hipMemcpyKind::hipMemcpyDeviceToHost);
  
  checkHostMatrix(hostArray, pitchInBytes, height, width, "\t%d", "Host data");
  
  SAFE_DELETE_CUDA(deviceArray);
  SAFE_DELETE_ARRAY(hostArray);
}
