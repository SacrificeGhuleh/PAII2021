#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <timer.h>
#include <type_traits>
#include <benchmark.h>

#define COMP_TYPE int

constexpr size_t width = 20000; //cols
constexpr size_t height = 20000; // rows
constexpr size_t numberOfTests = 20;


constexpr size_t widthInBytes = width * sizeof(COMP_TYPE); //cols
constexpr size_t heightInBytes = height * sizeof(COMP_TYPE); // rows
constexpr size_t length = width * height;
constexpr size_t lengthInBytes = length * sizeof(COMP_TYPE);

constexpr size_t TPB = 8;

// Print matrices, if matrices are small
constexpr bool printMatrices = (length <= 15 * 15);

hipError_t err = hipError_t::hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

//void printSize(size_t size) {
//  float flSize;
//  if (size < 1024UL) {
//    printf("%zu B\n", size);
//    return;
//  } else if (size < 1024UL * 1024UL) {
//    flSize = size / 1024.f;
//    printf("%f kB\n", flSize);
//    return;
//  } else if (size < 1024UL * 1024UL * 1024UL) {
//    flSize = size / (1024.f * 1024.f);
//    printf("%f MB\n", flSize);
//    return;
//  } else {
//    flSize = size / (1024.f * 1024.f * 1024.f);
//    printf("%f GB\n", flSize);
//    return;
//  }
//}

template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
struct Mat {
public:
  typedef T Type;
  
  Mat(size_t rows, size_t cols) :
      rows_(rows),
      cols_(cols),
      rowsInBytes_(rows * sizeof(Type)),
      colsInBytes_(cols * sizeof(Type)),
      deviceArray_(nullptr),
      hostArray_(nullptr) {
//    printf("Constructor\n");
    // Allocate device data
    checkCudaErrors(hipMallocPitch((void **) &deviceArray_, &pitchInBytes_, widthInBytes, height));
    pitchInElements_ = pitchInBytes_ / sizeof(COMP_TYPE);
    
    printf("Pitch: %zu B (%zu items)\n", pitchInBytes_, pitchInElements_);
    
    // Allocate host data
    hostArray_ = static_cast<COMP_TYPE *>(::operator new(pitchInBytes_ * height));
  }
  
  void free()
  /*~Mat()*/ {
//    printf("Destructor\n");
    SAFE_DELETE_CUDA(deviceArray_);
    SAFE_DELETE_ARRAY(hostArray_);
  }
  
  __device__ inline size_t getRowIdx(dim3 blockDim, dim3 blockIdx, dim3 threadIdx) const { return blockDim.x * blockIdx.x + threadIdx.x; }
  
  __device__ inline size_t getColIdx(dim3 blockDim, dim3 blockIdx, dim3 threadIdx) const { return blockDim.y * blockIdx.y + threadIdx.y; }
  
  __device__ __host__ inline size_t getIdx(size_t row, size_t col) const { return row * pitchInElements_ + col; }
  
  __device__ __host__ bool inBounds(size_t row, size_t col) const { return (row < rows_) && (col < cols_); }
  
  __device__ inline Type &atDevice(dim3 blockDim, dim3 blockIdx, dim3 threadIdx) {
    const size_t row = getRowIdx(blockDim, blockIdx, threadIdx);
    const size_t col = getColIdx(blockDim, blockIdx, threadIdx);
    return atDevice(row, col);
  }
  
  __device__ inline Type &atDevice(size_t row, size_t col) {
//      assert((row < rows_) && (col < cols_))
    const size_t idx = row * pitchInElements_ + col;
    return atDevice(idx);
  }
  
  __device__ inline Type &atDevice(size_t idx) {
    return deviceArray_[idx];
  }
  
  __host__ inline Type &atHost(size_t row, size_t col) {
//      assert((row < rows_) && (col < cols_))
    const size_t idx = row * pitchInElements_ + col;
    return atHost(idx);
  }
  
  __host__ inline Type &atHost(size_t idx) {
    return hostArray_[idx];
  }
  
  // Getters
  __device__ __host__ inline size_t getRows() const {
    return rows_;
  }
  
  __host__ inline void download() {
    hipMemcpy2D(hostArray_, pitchInBytes_, deviceArray_, pitchInBytes_, widthInBytes, height, hipMemcpyKind::hipMemcpyDeviceToHost);
  }
  
  __host__ inline void upload() {
    hipMemcpy2D(deviceArray_, pitchInBytes_, hostArray_, pitchInBytes_, widthInBytes, height, hipMemcpyKind::hipMemcpyHostToDevice);
  }
  
  __host__ inline void checkDeviceMatrix(const char *format = "%f ", const char *message = "") {
    ::checkDeviceMatrix(deviceArray_, pitchInBytes_, height, width, format, message);
  }
  
  __host__ inline void checkDeviceMatrix(bool uploadToDevice, const char *format = "%f ", const char *message = "") {
    if (uploadToDevice)
      upload();
    checkDeviceMatrix(format, message);
  }
  
  __host__ inline void checkHostMatrix(const char *format = "%f ", const char *message = "") {
    ::checkHostMatrix(hostArray_, pitchInBytes_, height, width, format, message);
  }
  
  __host__ inline void checkHostMatrix(bool downloadFromDevice, const char *format = "%f ", const char *message = "") {
    if (downloadFromDevice)
      download();
    checkHostMatrix(format, message);
  }
  
  __device__ __host__ inline size_t getCols() const {
    return cols_;
  }
  
  __device__ __host__ inline size_t getRowsInBytes() const {
    return rowsInBytes_;
  }
  
  __device__ __host__ inline size_t getColsInBytes() const {
    return colsInBytes_;
  }
  
  __device__ inline Type *getDeviceArray() {
    return deviceArray_;
  }
  
  __host__ inline Type *getHostArray() {
    return hostArray_;
  }
  
  __device__ __host__ inline size_t getPitchInBytes() const {
    return pitchInBytes_;
  }
  
  __device__ __host__ inline size_t getPitchInElements() const {
    return pitchInElements_;
  }

private:
  const size_t rows_;
  const size_t cols_;
  
  const size_t rowsInBytes_;
  const size_t colsInBytes_;
  
  Type *deviceArray_;
  Type *hostArray_;
  
  size_t pitchInBytes_;
  size_t pitchInElements_;
};

template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void fill(T *__restrict__ arr, const size_t pitchInElements) {
  const size_t col = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t row = blockDim.x * blockIdx.x + threadIdx.x;
  
  const size_t idx = row * pitchInElements + col;

//  printf("--------------------------\n");
//  printf("col: %d, row: %d idx: %d\n", col, row, idx);
//  printf("bd: %d, %d %d\n", blockDim.x, blockDim.y, blockDim.z);
//  printf("bi: %d, %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
//  printf("ti: %d, %d %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
//  printf("--------------------------\n\n");

//  printf("--------------------------\ncol: %d, row: %d idx: %d\nbd: %d, %d %d\nbi: %d, %d %d\nti: %d, %d %d\n--------------------------\n\n",
//         col, row, idx,
//         blockDim.x, blockDim.y, blockDim.z,
//         blockIdx.x, blockIdx.y, blockIdx.z,
//         threadIdx.x, threadIdx.y, threadIdx.z);
//
  
  if ((row < height) && (col < width)) {
    arr[idx] = col * height + row;
  }
}


template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void increment(T *__restrict__ arr, const size_t pitchInElements) {
  const size_t col = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t row = blockDim.x * blockIdx.x + threadIdx.x;
  
  const size_t idx = row * pitchInElements + col;
  
  if ((row < height) && (col < width)) {
//    arr[idx] = arr[idx] << 1;
    arr[idx] += 1;
  }
}


template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void fillMat(Mat<T> arr) {
  const size_t row = arr.getRowIdx(blockDim, blockIdx, threadIdx);
  const size_t col = arr.getColIdx(blockDim, blockIdx, threadIdx);
  if (arr.inBounds(row, col)) {
    arr.atDevice(row, col) = col * arr.getRows() + row;
  }
}


template<typename T, typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void incrementMat(Mat<T> arr) {
  const size_t row = arr.getRowIdx(blockDim, blockIdx, threadIdx);
  const size_t col = arr.getColIdx(blockDim, blockIdx, threadIdx);
  if (arr.inBounds(row, col)) {
    arr.atDevice(row, col)++;
//    arr.atDevice(row, col) = arr.atDevice(row, col) << 1;
  }
}

void testMat(int nrOfTests) {
  printf(" --- \n MATRIX\n --- \n");
  Mat<COMP_TYPE> myMat(height, width);
  
  printf("Pitch: %zu B (%zu items)\n", myMat.getPitchInBytes(), myMat.getPitchInElements());
  
  dim3 dimBlock(TPB, TPB, 1);
  dim3 dimGrid(getNumberOfParts(height, TPB), getNumberOfParts(width, TPB), 1);
  
  printf("Dim block: %d, %d, %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
  printf("Dim  grid: %d, %d, %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
  
  if (printMatrices) {
    //If printing is enabled, print filled result before benchmark
    
    fillMat<<<dimGrid, dimBlock>>>(myMat);
    printLastCudaError("ERROR: ");
    checkCudaErrors(hipDeviceSynchronize());
    
    myMat.checkDeviceMatrix("%4d ", "Matrix filled data");
  }
  
  auto testFunc = [&]() {
    
    fillMat<<<dimGrid, dimBlock>>>(myMat);
    printLastCudaError("ERROR: ");
    checkCudaErrors(hipDeviceSynchronize());
    
    incrementMat<<<dimGrid, dimBlock>>>(myMat);
    printLastCudaError("ERROR: ");
    checkCudaErrors(hipDeviceSynchronize());
    
  };
  
  gpubenchmark::print_time("My mat test", testFunc, nrOfTests);
  
  if (printMatrices) {
    myMat.checkHostMatrix(true, "%4d ", "Matrix incremented data");
  }
  
  myMat.free();
}

void testNative(int nrOfTests) {
  printf(" --- \n NATIVE\n --- \n");
  
  COMP_TYPE *deviceArray = nullptr;
  size_t pitchInBytes = 0;
  checkCudaErrors(hipMallocPitch((void **) &deviceArray, &pitchInBytes, widthInBytes, height));
  const size_t pitchInElements = pitchInBytes / sizeof(COMP_TYPE);
  
  printf("Pitch: %zu B (%zu items)\n", pitchInBytes, pitchInElements);
  
  dim3 dimBlock(TPB, TPB, 1);
  dim3 dimGrid(getNumberOfParts(height, TPB), getNumberOfParts(width, TPB), 1);
  
  printf("Dim block: %d, %d, %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
  printf("Dim  grid: %d, %d, %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
  
  if (printMatrices) {
    //If printing is enabled, print filled result before benchmark
    
    fill<<<dimGrid, dimBlock>>>(deviceArray, pitchInElements);
    printLastCudaError("ERROR: ");
    checkCudaErrors(hipDeviceSynchronize());
    
    checkDeviceMatrix(deviceArray, pitchInBytes, height, width, "%4d ", "Native filled data");
  }
  
  auto testFunc = [&]() {
    
    fill<<<dimGrid, dimBlock>>>(deviceArray, pitchInElements);
    printLastCudaError("ERROR: ");
    checkCudaErrors(hipDeviceSynchronize());
    
    increment<<<dimGrid, dimBlock>>>(deviceArray, pitchInElements);
    printLastCudaError("ERROR: ");
    checkCudaErrors(hipDeviceSynchronize());
    
  };
  
  gpubenchmark::print_time("Native test", testFunc, nrOfTests);
  
  COMP_TYPE *hostArray = static_cast<COMP_TYPE *>(::operator new(pitchInBytes * height));
  if (printMatrices) {
    hipMemcpy2D(hostArray, pitchInBytes, deviceArray, pitchInBytes, widthInBytes, height, hipMemcpyKind::hipMemcpyDeviceToHost);
    checkHostMatrix(hostArray, pitchInBytes, height, width, "%4d ", "Native incremented data");
  }
  
  SAFE_DELETE_CUDA(deviceArray);
  SAFE_DELETE_ARRAY(hostArray);
}

int main() {
  initializeCUDA(deviceProp);
  
  printf("testing %zu x %zu matrices\n", width, height);
//  printSize(length * sizeof(lengthInBytes));
  
  
  testNative(numberOfTests);
  testMat(numberOfTests);
}
